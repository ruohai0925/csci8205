extern "C" {
  #include "cuda_code.h"
}


#include <hip/hip_runtime.h>
// CUDA-C includes
#include <hip/hip_runtime.h>

int dev_id[4];
uint32_t num_dev = 0;

void device_info(){
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("Device %d: \"%s\"\n", dev, deviceProp.name);
        printf("  Total amount of global memory: %.0f MBytes (%llu bytes)\n",(float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
    }
}

void init_gpu_devices(){
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  int dev, driverVersion = 0, runtimeVersion = 0;
  for(dev = 0; dev < deviceCount; ++dev){
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    if((float)deviceProp.totalGlobalMem/1048576.0f > 6000){
      dev_id[num_dev] = dev;
      num_dev++;
      //printf("Device %d: \"%s\"\n", dev, deviceProp.name);
      //printf("  Total amount of global memory: %.0f MBytes (%llu bytes)\n",(float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
    }
    
  }


}